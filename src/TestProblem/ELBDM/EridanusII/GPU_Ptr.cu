
#ifdef __HIPCC__
#include "CUAPI.h"



extern int    Eridanus_Prof_NBin;
       
       real *d_ExtPotGrep_Eri;
extern void **d_ExtPotGenePtr;

//--------------------------------------------------------------------------
//Function:
//Description:
//Note:
//-------------------------------------------------------------------------
__host__
void SetGPUPtr(const real *h_table)
{
//   for ( int b = 0; b<3*Eridanus_Prof_NBin; b++)
//   {
//       printf("h table = %.4f\n", h_table[b]);
//   }
     
   const long MemSize = sizeof(real)*Eridanus_Prof_NBin*3;
 
   CUDA_CHECK_ERROR( hipMalloc((void**) &d_ExtPotGrep_Eri, MemSize));
//   real *h_00 = (real*)malloc(MemSize);
//   memset(h_00, 0, MemSize);
//   hipMemcpy(d_ExtPotGrep_Eri,h_00,MemSize,hipMemcpyHostToDevice);
  
//   printf("1\n");
   CUDA_CHECK_ERROR( hipMemcpy(d_ExtPotGrep_Eri, h_table,MemSize,hipMemcpyHostToDevice));
//   printf("2\n");
   real *d_ExtPotGrep_Ptr[3] = {d_ExtPotGrep_Eri,d_ExtPotGrep_Eri+Eridanus_Prof_NBin,d_ExtPotGrep_Eri+2*Eridanus_Prof_NBin};
//   printf("3\n");

   CUDA_CHECK_ERROR( hipMemcpy(d_ExtPotGenePtr, d_ExtPotGrep_Ptr, sizeof(real*)*3,hipMemcpyHostToDevice));
//   printf("4\n");
}

#endif




